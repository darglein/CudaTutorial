#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "tiny-eigen/matrix.h"

#include <iostream>
#include <vector>

#include <thrust/device_vector.h>

using vec3 = Eigen::Matrix<float, 3, 1>;

struct Particle
{
    vec3 position;
    float radius;
    vec3 velocity;
    float invMass;
};

// ===== Helper functions ====
__device__ inline int GlobalThreadId()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}
inline int iDivUp(int a, int b)
{
    return (a + b - (1)) / b;
}

// ===== Particle Integration Kernel ====
__global__ static void updateParticles(Particle* particles, int N, float dt)
{
    int tid = GlobalThreadId();
    if (tid >= N) return;
    Particle& p = particles[tid];
    p.position += p.velocity * dt;
    p.velocity += vec3(0, -9.81, 0) * dt;
}

int main(int argc, char* argv[])
{
    const int N     = 25;
    const int steps = 3;
    float dt        = 0.1;

    // Allocate CPU and GPU memory
    std::vector<Particle> particles(N);
    thrust::device_vector<Particle> d_particles(N);

    auto rand_float = []() { return ((rand() % 10000) / 10000.f) * 2 - 1; };

    // Initialize on the CPU
    for (Particle& p : particles)
    {
        p.position = vec3::Zero();
        p.velocity = vec3(rand_float(), rand_float(), rand_float());
    }

    // Upload memory
    thrust::copy(particles.begin(), particles.end(), d_particles.begin());

    // Integrate
    for (int i = 0; i < steps; ++i)
    {
        const int BLOCK_SIZE = 128;
        updateParticles<<<iDivUp(N, BLOCK_SIZE), BLOCK_SIZE>>>(
            d_particles.data().get(), N, dt);
    }

    // Download
    thrust::copy(d_particles.begin(), d_particles.end(), particles.begin());

    for (Particle& p : particles)
    {
        std::cout << p.position.transpose() << " " << p.velocity.transpose()
                  << std::endl;
    }
    std::cout << "done." << std::endl;
}
