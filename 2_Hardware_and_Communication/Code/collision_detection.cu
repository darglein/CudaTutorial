#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "Eigen/Core"
#include "Timer.h"

#include <iostream>

#include <thrust/device_vector.h>

const int max_collisions = 100000;
const int shared_factor  = 4;

__device__ int GlobalThreadId()
{
    return blockIdx.x * gridDim.x + threadIdx.x;
}
int iDivUp(int a, int b)
{
    return (a + b - (1)) / b;
}

using vec3 = Eigen::Vector3f;
struct Particle
{
    vec3 position;
    float radius;
};

__host__ __device__ bool Collide(const Particle& p1, const Particle& p2)
{
    float r2 = p1.radius + p2.radius;
    return (p1.position - p2.position).squaredNorm() < r2 * r2;
}

__global__ void RedBlueParticleCollision(Particle* particles1, Particle* particles2, int n, int m, int2* list,
                                         int* counter)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= n || j >= m) return;


    const Particle& p1 = particles1[i];
    const Particle& p2 = particles2[j];


    if (Collide(p1, p2))
    {
        int index = atomicAdd(counter, 1);
        if (index < max_collisions)
        {
            list[index] = {i, j};
        }
    }
}

template <int BLOCK_SIZE_X, int BLOCK_SIZE_Y>
__global__ void RedBlueParticleCollisionShared(Particle* particles1, Particle* particles2, int n, int m, int2* list,
                                               int* counter)
{
    __shared__ float4 shared_particles1[BLOCK_SIZE_X * shared_factor];
    __shared__ float4 shared_particles2[BLOCK_SIZE_Y * shared_factor];


    int block_i = blockIdx.x * blockDim.x * shared_factor;
    int block_j = blockIdx.y * blockDim.y * shared_factor;

    if (block_i >= n || block_j >= m) return;

    // Load to smem
    if (threadIdx.y < shared_factor)
    {
        static_assert(shared_factor <= BLOCK_SIZE_X && shared_factor <= BLOCK_SIZE_Y, "a");
        int offset                = threadIdx.x + threadIdx.y * 16;
        shared_particles1[offset] = ((float4*)particles1)[block_i + offset];
        shared_particles2[offset] = ((float4*)particles2)[block_j + offset];
    }



    __syncthreads();

    for (int k = 0; k < shared_factor; ++k)
    {
        for (int l = 0; l < shared_factor; ++l)
        {
            int local_i = k * blockDim.x + threadIdx.x;
            int local_j = l * blockDim.y + threadIdx.y;

            int i = block_i + local_i;
            int j = block_j + local_j;

            if (i >= n || j >= m) continue;



            const Particle& p1 = ((Particle*)shared_particles1)[local_i];
            const Particle& p2 = ((Particle*)shared_particles2)[local_j];


            if (Collide(p1, p2))
            {
                int index = atomicAdd(counter, 1);
                if (index < max_collisions)
                {
                    list[index] = {i, j};
                }
            }
        }
    }
}

int main(int argc, char* argv[])
{
    int n = 5000;
    int m = 5000;

    const int block_size_x = 16;
    const int block_size_y = 16;

    std::vector<Particle> particles1(n);
    std::vector<Particle> particles2(m);

    srand(1056735);
    for (Particle& p : particles1)
    {
        p.position = vec3::Random() * 10;
        p.radius   = 1;
    }
    for (Particle& p : particles2)
    {
        p.position = vec3::Random() * 10;
        p.radius   = 1;
    }

    thrust::device_vector<Particle> d_particles1(particles1);
    thrust::device_vector<Particle> d_particles2(particles2);

    // Add padding to simplify shared memory kernel
    d_particles1.resize(n + 16 * shared_factor);
    d_particles2.resize(m + 16 * shared_factor);

    thrust::device_vector<int2> d_collision_list(max_collisions);
    thrust::device_vector<int> d_collision_count(1, 0);
    d_collision_count[0] = 0;


    float time_gpu1, time_gpu2;
    {
        int blocks_x = iDivUp(n, block_size_x);
        int blocks_y = iDivUp(m, block_size_y);
        CudaScopedTimer timer(time_gpu1);
        RedBlueParticleCollision<<<dim3(blocks_x, blocks_y, 1), dim3(block_size_x, block_size_y, 1)>>>(
            d_particles1.data().get(), d_particles2.data().get(), n, m, d_collision_list.data().get(),
            d_collision_count.data().get());
    }

    int num_collisions = d_collision_count[0];
    std::cout << "Found " << num_collisions << " collisions on the GPU in " << time_gpu1 << " ms" << std::endl;
    d_collision_count[0] = 0;
    {
        int blocks_x = iDivUp(n, block_size_x * shared_factor);
        int blocks_y = iDivUp(m, block_size_y * shared_factor);

        CudaScopedTimer timer(time_gpu2);
        RedBlueParticleCollisionShared<block_size_x, block_size_y>
            <<<dim3(blocks_x, blocks_y, 1), dim3(block_size_x, block_size_y, 1)>>>(
                d_particles1.data().get(), d_particles2.data().get(), n, m, d_collision_list.data().get(),
                d_collision_count.data().get());
    }
    hipDeviceSynchronize();

    num_collisions = d_collision_count[0];

    std::cout << "Found " << num_collisions << " collisions on the GPU in " << time_gpu2 << " ms" << std::endl;

    //    return 0;
    std::atomic_int num_collisions_cpu;
    num_collisions_cpu = 0;
    float time_cpu;
    {
        ScopedTimer<float> timer(time_cpu);
#pragma omp parallel for
        for (int i = 0; i < particles1.size(); ++i)
        {
            auto& p1 = particles1[i];
            for (auto& p2 : particles2)
            {
                if (Collide(p1, p2))
                {
                    num_collisions_cpu++;
                }
            }
        }
    }

    std::cout << "Found " << num_collisions_cpu << " collisions on the CPU in " << time_cpu << " ms" << std::endl;
    return 0;
}
