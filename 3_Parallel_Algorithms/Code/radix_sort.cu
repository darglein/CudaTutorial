#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */


#include <iostream>
#include <algorithm>

#include "bitonic_sort.h"
#include <thrust/device_vector.h>


template <bool one>
struct GetBitOp
{
    int k;
    GetBitOp(int k) : k(k) {}
    __device__ inline int operator()(int a) { return ((a >> k) & 1) == one; }
};

static void radixSortHelper(thrust::device_vector<int>& d, thrust::device_vector<int>& p, thrust::device_vector<int>& s,
                            thrust::device_vector<int>& t, int bit)
{
#if 1
    // Implementation with scan+scatter

    // Compute predicate array for 0-bits
    thrust::transform(d.begin(), d.end(), p.begin(), GetBitOp<false>(bit));

    // Scan over the predicate array and store it in s
    thrust::exclusive_scan(p.begin(), p.end(), s.begin(), 0);

    // Write all 0-bit integers to the scanned positions
    // This writes only if the predicate also evaluates to true
    thrust::scatter_if(d.begin(), d.end(), s.begin(), p.begin(), t.begin());

    // Total number of 0 bits
    int count = thrust::reduce(p.begin(), p.end());

    // Same with 1-bit integers, but use 'count' as the initial value in the scan
    thrust::transform(d.begin(), d.end(), p.begin(), GetBitOp<true>(bit));
    thrust::exclusive_scan(p.begin(), p.end(), s.begin(), count);
    thrust::scatter_if(d.begin(), d.end(), s.begin(), p.begin(), t.begin());
#else
    // Implementation with copy_if
    auto it = thrust::copy_if(d.begin(), d.end(), t.begin(), GetBitOp<false>(bit));
    thrust::copy_if(d.begin(), d.end(), it, GetBitOp<true>(bit));
#endif

    // Both variants don't work inplace!
    thrust::copy(t.begin(), t.end(), d.begin());
}

static void radixSort(thrust::device_vector<int>& data)
{
    int N = data.size();

    // Temporary arrays
    thrust::device_vector<int> pred(N);
    thrust::device_vector<int> scan(N);
    thrust::device_vector<int> temp(N);

    // Sort from least to most significant bit
    for (int i = 0; i < 32; ++i) radixSortHelper(data, pred, scan, temp, i);
}

int main(int argc, char* argv[])
{
    int N   = 1024 * 1024;
    using T = int;
    thrust::host_vector<T> h_data(N);

    // Initialize with random values
    for (auto& f : h_data)
    {
        f = abs(rand());
    }


    std::cout << "Sorting " << N << " elements..." << std::endl;
    thrust::device_vector<T> d_data = h_data;
    radixSort(d_data);

    thrust::host_vector<T> res = d_data;
    if (std::is_sorted(res.begin(), res.end()))
    {
        std::cout << "Success! All elements are in the correct order!" << std::endl;
    }
    else
    {
        std::cout << "Sort failed!" << std::endl;
    }

    return 0;
}
