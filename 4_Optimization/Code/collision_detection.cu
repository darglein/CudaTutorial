#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "Timer.h"

#include <iostream>

#include "tiny-eigen/matrix.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>

using vec3               = Eigen::Matrix<float, 3, 1>;
const int MAX_COLLISIONS = 100000;

__device__ int GlobalThreadId()
{
    return blockIdx.x * gridDim.x + threadIdx.x;
}
int iDivUp(int a, int b)
{
    return (a + b - (1)) / b;
}

struct Particle
{
    vec3 position;
    float radius;
};

__host__ __device__ bool Collide(const Particle& p1, const Particle& p2)
{
    float r2 = p1.radius + p2.radius;
    return (p1.position - p2.position).squaredNorm() < r2 * r2;
}

// =======================================================================================

__global__ void RedBlueParticleCollisionSimple(Particle* particles1, Particle* particles2, int n, int m, int2* list,
                                               int* counter)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= n || j >= m) return;


    const Particle& p1 = particles1[i];
    const Particle& p2 = particles2[j];


    if (Collide(p1, p2))
    {
        int index = atomicAdd(counter, 1);
        if (index < MAX_COLLISIONS)
        {
            list[index] = {i, j};
        }
    }
}


template <int BLOCK_SIZE_X, int BLOCK_SIZE_Y, int K>
__global__ void RedBlueParticleCollisionShared(Particle* particles1, Particle* particles2, int n, int m, int2* list,
                                               int* counter)
{
    __shared__ Particle shared_particles1[16 * K];
    __shared__ Particle shared_particles2[16 * K];

    int block_i = blockIdx.x * 16 * K;
    int block_j = blockIdx.y * 16 * K;
    // int tid     = threadIdx.x + threadIdx.y * blockDim.x;

    if (block_i >= n || block_j >= m) return;

    // Load to smem
    if (threadIdx.y < K)
    {
        static_assert(K <= BLOCK_SIZE_X && K <= BLOCK_SIZE_Y, "a");
        int offset                = threadIdx.x + threadIdx.y * 16;
        shared_particles1[offset] = (particles1)[block_i + offset];
        shared_particles2[offset] = (particles2)[block_j + offset];
    }

    __syncthreads();

    for (int k = 0; k < K; ++k)
    {
        for (int l = 0; l < K; ++l)
        {
            int local_i = k * blockDim.x + threadIdx.x;
            int local_j = l * blockDim.y + threadIdx.y;

            int i = block_i + local_i;
            int j = block_j + local_j;

            if (i >= n || j >= m) continue;

            const Particle& p1 = shared_particles1[local_i];
            const Particle& p2 = shared_particles2[local_j];

            if (Collide(p1, p2))
            {
                int index = atomicAdd(counter, 1);
                if (index < MAX_COLLISIONS)
                {
                    list[index] = {i, j};
                }
            }
        }
    }
}

template <int BLOCK_SIZE_X, int BLOCK_SIZE_Y, int K>
__global__ void RedBlueParticleCollisionSharedOptimized(Particle* particles1, Particle* particles2, int n, int m,
                                                        int2* list, int* counter)
{
    __shared__ float4 shared_particles1[16 * K];
    __shared__ float4 shared_particles2[16 * K];


    int block_i = blockIdx.x * 16 * K;
    int block_j = blockIdx.y * 16 * K;
    //    int tid     = threadIdx.x + threadIdx.y * blockDim.x;

    if (block_i >= n || block_j >= m) return;

    // Load to smem
    if (threadIdx.y < K)
    {
        static_assert(K <= BLOCK_SIZE_X && K <= BLOCK_SIZE_Y, "a");
        int offset                = threadIdx.x + threadIdx.y * 16;
        shared_particles1[offset] = ((float4*)particles1)[block_i + offset];
        shared_particles2[offset] = ((float4*)particles2)[block_j + offset];
    }



    __syncthreads();


    for (int k = 0; k < K * (16 / BLOCK_SIZE_X); ++k)
    {
        int local_i = k * blockDim.x + threadIdx.x;
        int i       = block_i + local_i;
        Particle p1 = ((Particle*)shared_particles1)[local_i];

        for (int l = 0; l < K * (16 / BLOCK_SIZE_Y); ++l)
        {
            int local_j = l * blockDim.y + threadIdx.y;
            int j       = block_j + local_j;
            Particle p2 = ((Particle*)shared_particles2)[local_j];
            if (j >= m | i >= n) p2.position.x() = std::numeric_limits<float>::quiet_NaN();


            if (Collide(p1, p2))
            {
                int index = atomicAdd(counter, 1);
                if (index < MAX_COLLISIONS)
                {
                    list[index] = {i, j};
                }
            }
        }
    }
}

int main(int argc, char* argv[])
{
    int n = 3940;
    int m = 5824;

    const int K     = 4;
    const int steps = 11;

    const int block_size_x = 16;
    const int block_size_y = 16;

    std::vector<Particle> particles1(n);
    std::vector<Particle> particles2(m);

    srand(1056735);
    auto rand_float = []() { return ((rand() % 10000) / 10000.f) * 2 - 1; };
    for (Particle& p : particles1)
    {
        p.position = vec3(rand_float(), rand_float(), rand_float()) * 25;
        p.radius   = 1;
    }
    for (Particle& p : particles2)
    {
        p.position = vec3(rand_float(), rand_float(), rand_float()) * 25;
        p.radius   = 1;
    }

    thrust::device_vector<Particle> d_particles1(particles1);
    thrust::device_vector<Particle> d_particles2(particles2);

    // Add padding to simplify shared memory kernel
    d_particles1.resize(n + 16 * K);
    d_particles2.resize(m + 16 * K);

    thrust::device_vector<int2> d_collision_list(MAX_COLLISIONS);
    thrust::device_vector<int> d_collision_count(1, 0);
    d_collision_count[0] = 0;


    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    hipProfilerStart();


    float time_simple = measureObject2(
        steps, [&]() { d_collision_count[0] = 0; },
        [&]()
        {
            int blocks_x = iDivUp(n, block_size_x);
            int blocks_y = iDivUp(m, block_size_y);
            RedBlueParticleCollisionSimple<<<dim3(blocks_x, blocks_y, 1), dim3(block_size_x, block_size_y, 1)>>>(
                d_particles1.data().get(), d_particles2.data().get(), n, m, d_collision_list.data().get(),
                d_collision_count.data().get());
        });
    std::cout << "RedBlueParticleCollisionSimple            " << time_simple << " ms." << std::endl;
    int num_collisions_simple = d_collision_count[0];

#if 1
    float time_shared = measureObject2(
        steps, [&]() { d_collision_count[0] = 0; },
        [&]()
        {
            int blocks_x = iDivUp(n, block_size_x * K);
            int blocks_y = iDivUp(m, block_size_y * K);
            RedBlueParticleCollisionShared<block_size_x, block_size_y, K>
                <<<dim3(blocks_x, blocks_y, 1), dim3(block_size_x, block_size_y, 1)>>>(
                    d_particles1.data().get(), d_particles2.data().get(), n, m, d_collision_list.data().get(),
                    d_collision_count.data().get());
        });
    std::cout << "RedBlueParticleCollisionShared            " << time_shared << " ms." << std::endl;
#endif
    int num_collisions_shared = d_collision_count[0];

#if 1
    float time_shared_optimized = measureObject2(
        steps, [&]() { d_collision_count[0] = 0; },
        [&]()
        {
            int blocks_x = iDivUp(n, block_size_x * K);
            int blocks_y = iDivUp(m, block_size_y * K);
            RedBlueParticleCollisionSharedOptimized<block_size_x, block_size_y, K>
                <<<dim3(blocks_x, blocks_y, 1), dim3(block_size_x, block_size_y, 1)>>>(
                    d_particles1.data().get(), d_particles2.data().get(), n, m, d_collision_list.data().get(),
                    d_collision_count.data().get());
        });
    std::cout << "RedBlueParticleCollisionSharedOptimized   " << time_shared_optimized << " ms." << std::endl;
#endif
    int num_collisions_shared_optimized = d_collision_count[0];

    hipProfilerStop();
    hipDeviceSynchronize();

    std::atomic_int num_collisions_cpu;
    num_collisions_cpu = 0;
    float time_cpu;
    {
        ScopedTimer<float> timer(time_cpu);
#pragma omp parallel for
        for (int i = 0; i < particles1.size(); ++i)
        {
            auto& p1 = particles1[i];
            for (auto& p2 : particles2)
            {
                if (Collide(p1, p2))
                {
                    num_collisions_cpu++;
                }
            }
        }
    }
    std::cout << "RedBlueParticleCollision CPU              " << time_cpu << " ms" << std::endl;
    std::cout << std::endl;
    std::cout << "Num Collisions GPU: " << num_collisions_simple << " " << num_collisions_shared << " "
              << num_collisions_shared_optimized << std::endl;
    std::cout << "Num Collisions CPU: " << num_collisions_cpu << std::endl;

    return 0;
}
